#include "hip/hip_runtime.h"
#include "render.h"

namespace FaceModeling {

template<typename T>
__device__ void Normal_base_f(const T* normal,
                            T* normal_b)
{
    normal_b[0] = T(1);
    normal_b[1] = normal[1];
    normal_b[2] = normal[2];
    normal_b[3] = normal[0];
    normal_b[4] = normal[0] * normal[1];
    normal_b[5] = normal[1] * normal[2];
    normal_b[6] = T(3)*normal[2]*normal[2] - T(1);
    normal_b[7] = normal[0]*normal[2];
    normal_b[8] = normal[0]*normal[0] - normal[1]*normal[1];
}

template<typename T>
__device__ void Syn_color_f(const T* normal_base, const T* diffuse, const T* texture,
                          T* syn_c)
{
    for(int channel=0; channel<3; channel++)
    {
        T sum = T(0);
        for(int i=0; i<9 ;i++)
        {
            sum = sum+normal_base[i]*diffuse[9*channel+i];
        }
        syn_c[channel] = texture[channel]*sum;
    }
}

template <typename T>
__global__ void rendermesh(int nthread, const T* ori_img, const pixel_info* pixel_gpu, const int* inds, const T* texture,
                           const T* diffuse_sh, unsigned char* dimg, int width, int height)
{
    CUDA_KERNEL_LOOP(index, nthread)
    {
        int yy = index/width;
        int xx = index%width;
        for(int c=0; c<3; c++)
        {
            dimg[3*(width*yy+xx)+2-c] = ori_img[c*width*height+yy*width+xx];
        }

        int tri_ind = pixel_gpu[index].tri_index;
        if(tri_ind != -1)
        {
            if(inds[tri_ind] == 1)
            {
                T syn_color[3];
                int x = pixel_gpu[index].x;
                int y = pixel_gpu[index].y;
                T normal_base[9];
                Normal_base_f(pixel_gpu[index].normal, normal_base);
                Syn_color_f(normal_base, diffuse_sh, texture, syn_color);
                for(int c=0; c<3; c++)
                {
                    T rend_c = 0.0*syn_color[2-c] + 1.0*dimg[3*(width*(height-1-y)+x)+c];
                    if(rend_c < 0)
                    {
                        rend_c = 0;
                    }
                    if(rend_c > 255)
                    {
                        rend_c = 255;
                    }
                    dimg[3*(width*(height-1-y)+x)+c] = (unsigned char)rend_c;
                }
            }
        }
    }
}

template <typename T>
__global__ void rendermesh_uchar(int nthread, const unsigned char* ori_img, const pixel_info* pixel_gpu, const int* inds, const T* texture,
                           const T* diffuse_sh, unsigned char* dimg, int width, int height)
{
    CUDA_KERNEL_LOOP(index, nthread)
    {
        int yy = index/width;
        int xx = index%width;
        for(int c=0; c<3; c++)
        {
            dimg[3*(width*yy+xx)+2-c] = ori_img[3*(width*yy+xx)+2-c];
        }

        int tri_ind = pixel_gpu[index].tri_index;
        if(tri_ind != -1)
        {
            if(inds[tri_ind] == 1)
            {
                T syn_color[3];
                int x = pixel_gpu[index].x;
                int y = pixel_gpu[index].y;
                T normal_base[9];
                Normal_base_f(pixel_gpu[index].normal, normal_base);
                Syn_color_f(normal_base, diffuse_sh, texture, syn_color);
                for(int c=0; c<3; c++)
                {
                    T rend_c = 1.0*syn_color[2-c] + 0.0 *dimg[3*(width*(height-1-y)+x)+c];
                    if(rend_c < 0)
                    {
                       rend_c = 0;
                    }
                    if(rend_c > 255)
                    {
                        rend_c = 255;
                    }
                    dimg[3*(width*(height-1-y)+x)+c] = (unsigned char)rend_c;
                }
            }
        }
    }
}


template <typename T>
__global__ void render_3dmm(int nthread, const unsigned char* ori_img, const pixel_info* pixel_gpu,
                            const int* inds, const T* texture, const T* diffuse_sh,
                            unsigned char* dimg, int width, int height)
{
    CUDA_KERNEL_LOOP(index, nthread)
    {
        int yy = index/width;
        int xx = index%width;
        for(int c=0; c<3; c++)
        {
            dimg[3*(width*yy+xx)+2-c] = ori_img[3*(width*yy+xx)+2-c];
        }

        int tri_ind = pixel_gpu[index].tri_index;
        if(tri_ind != -1)
        {
            if(inds[tri_ind] == 1)
            {
                int vinds[3];
                vinds[0] = pixel_gpu[index].v1index;
                vinds[1] = pixel_gpu[index].v2index;
                vinds[2] = pixel_gpu[index].v3index;
                T syn_color[3];
                int x = pixel_gpu[index].x;
                int y = pixel_gpu[index].y;
                T normal_base[9], tex[3];
                for(int c=0; c<3; c++)
                {
                    tex[c] = 0;
                    for(int j=0; j<3; j++)
                    {
                        tex[c] += pixel_gpu[index].coord[j]*texture[3*vinds[j]+c];
                    }
                }
                Normal_base_f(pixel_gpu[index].normal, normal_base);
                Syn_color_f(normal_base, diffuse_sh, tex, syn_color);
                for(int c=0; c<3; c++)
                {
                    T rend_c = 1.0*syn_color[2-c] + 0.0 *dimg[3*(width*(height-1-y)+x)+c];
                    if(rend_c < 0)
                    {
                       rend_c = 0;
                    }
                    if(rend_c > 255)
                    {
                        rend_c = 255;
                    }
                    dimg[3*(width*(height-1-y)+x)+c] = (unsigned char)rend_c;
                }
            }
        }
    }
}


void Render::render_mesh(const T* ori_img, const pixel_info *pixels, unsigned char *d_img, int width, int height)
{
    int nthread = width*height;
    T* d_tex;
    hipMalloc(&d_tex, sizeof(T)*3);
    float c_tex[3] = {125, 125, 125};
    hipMemcpy(d_tex, c_tex, sizeof(T)*3, hipMemcpyHostToDevice);
    rendermesh<<< FaceModeling_GET_BLOCKS(nthread), FaceModeling_CUDA_NUM_THREADS >>>
    (nthread, ori_img, pixels, d_ind_without_eye, d_tex, d_light, d_img, width, height);
    safefree(d_tex);
}

void Render::render_mesh(const unsigned char* ori_img, const pixel_info *pixels, unsigned char *d_img, int width, int height)
{
    int nthread = width*height;
    T* d_tex;
    hipMalloc(&d_tex, sizeof(T)*3);
    float c_tex[3] = {125, 125, 125};
    hipMemcpy(d_tex, c_tex, sizeof(T)*3, hipMemcpyHostToDevice);
    rendermesh_uchar<<< FaceModeling_GET_BLOCKS(nthread), FaceModeling_CUDA_NUM_THREADS >>>
    (nthread, ori_img, pixels, d_ind_without_eye, d_tex, d_light, d_img, width, height);
    safefree(d_tex);
}

void Render::render(const unsigned char *ori_img, const pixel_info *pixels, const T* paras, const T *texture,
                    unsigned char *d_img, int width, int height)
{
    int nthread = width*height;
    hipMemcpy(d_diffuse_light, paras+285, sizeof(T)*27, hipMemcpyDeviceToDevice);
    render_3dmm<<< FaceModeling_GET_BLOCKS(nthread), FaceModeling_CUDA_NUM_THREADS >>>
    (nthread, ori_img, pixels, d_ind_without_eye, texture, d_diffuse_light, d_img, width, height);
}

}  // namespace FaceModeling

